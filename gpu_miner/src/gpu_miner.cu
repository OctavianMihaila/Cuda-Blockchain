#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

__global__ void findNonce(uint32_t *nonce, BYTE *block_content, size_t current_length, BYTE *block_hash, BYTE *difficulty) {
    uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t step = gridDim.x * blockDim.x;

    BYTE new_block_content[BLOCK_SIZE];
    BYTE new_block_hash[SHA256_HASH_SIZE];
    char nonce_as_string[SHA256_HASH_SIZE];

    // Copy block content to new block content in order to prevent other threads
    // from modifying it while this thread is calculating the hash.
    // Causes performance issues, but I don't know how to fix it.
    memcpy(new_block_content, block_content, current_length); // BOTTLENECK

    for (uint32_t nonce_to_try = thread_id; nonce_to_try <= MAX_NONCE; nonce_to_try += step) {
        intToString(nonce_to_try, nonce_as_string);
        size_t nonce_length = d_strlen(nonce_as_string);

        // Append nonce to block content
        memcpy(new_block_content + current_length, nonce_as_string, nonce_length + 1);

        apply_sha256(new_block_content, current_length + nonce_length, new_block_hash, 1);
        
        // Also using new_block_hash specific to this thread in order to prevent other
        // threads from overwriting it while this thread is comparing the hashes.
        if (compare_hashes(new_block_hash, difficulty) <= 0) {
            if (nonce_to_try < *nonce || *nonce == 0) {
                // Save the nonce and block hash
                atomicExch(nonce, nonce_to_try);
                memcpy(block_hash, new_block_hash, SHA256_HASH_SIZE);
            }

            return;
        }
    }
}

int main(int argc, char **argv) {
    BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
            tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
            tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
    BYTE block_hash[SHA256_HASH_SIZE], *d_block_content, *d_block_hash, *d_difficulty;
    uint32_t nonce, *d_nonce;
    size_t current_length;

    hipMalloc((void**)&d_block_content, BLOCK_SIZE);
    hipMalloc((void**)&d_block_hash, SHA256_HASH_SIZE);
    hipMalloc((void**)&d_nonce, sizeof(uint32_t));
    hipMalloc((void**)&d_difficulty, SHA256_HASH_SIZE);

    // Top hash
    apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
    apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
    apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
    apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
    strcpy((char *)tx12, (const char *)hashed_tx1);
    strcat((char *)tx12, (const char *)hashed_tx2);
    apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
    strcpy((char *)tx34, (const char *)hashed_tx3);
    strcat((char *)tx34, (const char *)hashed_tx4);
    apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
    strcpy((char *)tx1234, (const char *)hashed_tx12);
    strcat((char *)tx1234, (const char *)hashed_tx34);
    apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

    // Prev_block_hash + top_hash
    strcpy((char*)block_content, (const char*)prev_block_hash);
    strcat((char*)block_content, (const char*)top_hash);
    current_length = strlen((char*) block_content);

    // Copy block content and difficulty to device
    hipMemcpy(d_block_content, block_content, BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_difficulty, difficulty_5_zeros, SHA256_HASH_SIZE, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    startTiming(&start, &stop);

    findNonce<<<112, 512>>>(d_nonce, d_block_content, current_length, d_block_hash, d_difficulty);

    hipDeviceSynchronize();

    // Copy the result back to host
    hipMemcpy(&nonce, d_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(block_hash, d_block_hash, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);

    float seconds = stopTiming(&start, &stop);
    printResult(block_hash, nonce, seconds);

    hipFree(d_block_content);
    hipFree(d_block_hash);
    hipFree(d_nonce);
    hipFree(d_difficulty);

    return 0;
}
